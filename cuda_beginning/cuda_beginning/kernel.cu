// CUDACast #2

#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE 1024

// Enable to launch on GPU
// tells compiler that function is going to be executed
// on the gpu and callable on the host
__global__ void VectorAdd(int* a, int* b, int* c, int n) {
	// Select the thread index to figure out how to index into vector 
	int i = threadIdx.x;
	if (i < n)
		c[i] = a[i] + b[i];
}

int main() {
	int* a, * b, * c;
	int* d_a, * d_b, * d_c;
	a = (int*)malloc(SIZE * sizeof(int));
	b = (int*)malloc(SIZE * sizeof(int));
	c = (int*)malloc(SIZE * sizeof(int));

	// Must allocate memory on GPU
	hipMalloc(&d_a, SIZE * sizeof(int));
	hipMalloc(&d_b, SIZE * sizeof(int));
	hipMalloc(&d_c, SIZE * sizeof(int));
	
	for (int i = 0; i < SIZE; i++) {
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	// Copies values to GPU
	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE * sizeof(int), hipMemcpyHostToDevice);

	// Launch VectorAdd kernel on GPU
	// <<<BLOCKS, #Threads>>>
	VectorAdd<<< 1, SIZE >>>(d_a, d_b, d_c, SIZE);

	// Copy from GPU back to CPU
	hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	// Check results
	for (int i = 0; i < 10; i++)
		printf("c[%d] = %d\n", i, c[i]);
	
	// Free from CPU
	free(a);
	free(b);
	free(c);

	// Free from cuda
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}